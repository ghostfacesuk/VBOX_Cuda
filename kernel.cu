#include "hip/hip_runtime.h"
﻿#include <algorithm>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <stdio.h>

// Define the struct for storing data
struct DataPoint {
    float sats;
    float time;
    float lat;
    float lon;
    float velocity;
    float heading;
    float height;
    float vertVel;
    float tsample;
    int solutionType;
    int avifileindex;
    int avitime;
    float pps_mim;
    float teensyCount;
};

// CUDA kernel to process data
__global__ void processData(DataPoint* data, int numPoints, int totalBlocks) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = totalBlocks * blockDim.x;

    for (int i = idx; i < numPoints; i += stride) {
        // Process data[i]
        printf("Thread ID: %d, Time: %.3f, Lat: %.8f, Lon: %.8f, Velocity: %.3f\n",
            i, data[i].time, data[i].lat, data[i].lon, data[i].velocity);
    }
}

int main() {
    // Read data from the .vbo file
    std::ifstream file("data.vbo");
    if (!file.is_open()) {
        std::cerr << "Failed to open file." << std::endl;
        return 1;
    }

    // Skip the column names
    std::string line;
    std::getline(file, line);

    // Read data points
    std::vector<DataPoint> dataPoints;
    DataPoint point;
    while (file >> point.sats >> point.time >> point.lat >> point.lon >> point.velocity >>
        point.heading >> point.height >> point.vertVel >> point.tsample >> point.solutionType >>
        point.avifileindex >> point.avitime >> point.pps_mim >> point.teensyCount) {
        dataPoints.push_back(point);
    }
    file.close();

    // Print debug information about the data points
    std::cout << "Number of data points: " << dataPoints.size() << std::endl;
    for (int i = 0; i < std::min(static_cast<int>(dataPoints.size()), 10); ++i) {
        std::cout << "Data point " << i << ": " << dataPoints[i].time << ", " << dataPoints[i].lat << ", " << dataPoints[i].lon << ", " << dataPoints[i].velocity << std::endl;
    }

    // Prepare data for GPU
    DataPoint* d_data;
    hipMalloc(&d_data, dataPoints.size() * sizeof(DataPoint));
    hipMemcpy(d_data, dataPoints.data(), dataPoints.size() * sizeof(DataPoint), hipMemcpyHostToDevice);

    // Define CUDA kernel configuration
    int blockSize = 128;
    const int MAX_BLOCKS = 65536;
    int numBlocks = (dataPoints.size() + blockSize - 1) / blockSize;
    if (numBlocks > MAX_BLOCKS) {
        numBlocks = MAX_BLOCKS;
    }
    int totalBlocks = (dataPoints.size() + blockSize * numBlocks - 1) / (blockSize * numBlocks);

    // Launch CUDA kernel to process data
    for (int i = 0; i < totalBlocks; ++i) {
        processData << <numBlocks, blockSize >> > (d_data, dataPoints.size(), totalBlocks);
        hipDeviceSynchronize();

        // Check for kernel launch errors
        hipError_t hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            std::cerr << "CUDA kernel launch error: " << hipGetErrorString(hipError_t) << std::endl;
            return 1;
        }

        // Check for kernel execution errors
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            std::cerr << "CUDA kernel execution error: " << hipGetErrorString(hipError_t) << std::endl;
            return 1;
        }
    }

    // Flush any buffered output from the kernel
    hipDeviceReset();

    // Clean up
    hipFree(d_data);

    return 0;
}